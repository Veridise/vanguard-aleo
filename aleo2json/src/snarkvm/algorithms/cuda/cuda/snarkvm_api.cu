#include "hip/hip_runtime.h"
// Copyright (C) 2019-2022 Aleo Systems Inc.
// This file is part of the snarkVM library.

// The snarkVM library is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.

// The snarkVM library is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
// GNU General Public License for more details.

// You should have received a copy of the GNU General Public License
// along with the snarkVM library. If not, see <https://www.gnu.org/licenses/>.

#include <hip/hip_runtime.h>

#include "snarkvm.cu"

#ifndef __CUDA_ARCH__

// Lazy instantiation of snarkvm_t
class snarkvm_singleton_t {
    bool failed = false;
    snarkvm_t *snarkvm = nullptr;

public:
    snarkvm_singleton_t() {}
    ~snarkvm_singleton_t() {
        delete snarkvm;
        snarkvm = nullptr;
    }
    bool ok() {
        if (!failed && snarkvm == nullptr) {
            // SNP TODO: max domain size?
            snarkvm = new snarkvm_t(17);
            if (snarkvm == nullptr) {
                failed = true;
            }
        }
        return snarkvm != nullptr;
    }
    snarkvm_t* operator->() {
        assert (ok());
        return snarkvm;
    }
};
snarkvm_singleton_t snarkvm_g;
                                         
#ifndef __CUDA_ARCH__

extern "C" {
    RustError snarkvm_ntt(fr_t* inout, uint32_t lg_domain_size,
                          NTT::InputOutputOrder ntt_order, NTT::Direction ntt_direction,
                          NTT::Type ntt_type)
    {
        if (!snarkvm_g.ok()) {
            return RustError{hipErrorOutOfMemory};
        }
        return snarkvm_g->NTT(inout, inout, lg_domain_size, ntt_order,
                              ntt_direction, ntt_type);
    }

    RustError snarkvm_polymul(fr_t* out,
                              size_t pcount, fr_t** polynomials, size_t* plens,
                              size_t ecount, fr_t** evaluations, size_t* elens,
                              uint32_t lg_domain_size) {
        if (!snarkvm_g.ok()) {
            return RustError{hipErrorOutOfMemory};
        }
        return snarkvm_g->PolyMul(out,
                                  pcount, polynomials, plens,
                                  ecount, evaluations, elens,
                                  lg_domain_size);
    }

    RustError snarkvm_msm(point_t* out, const affine_t points[], size_t npoints,
                          const scalar_t scalars[], size_t ffi_affine_size) {
        if (!snarkvm_g.ok()) {
            return RustError{hipErrorOutOfMemory};
        }
        return snarkvm_g->MSM(out, points, npoints, scalars, ffi_affine_size);
    }
}
#endif // __CUDA_ARCH__

#endif
